#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void matmul(int * a, int *b, int *res,int n)  // for multip.. of CD^T
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	int sum = 0;
	for(int k=0;k<n;k++)
	{
		sum += a[i*n+k]* b[j*n+k];
	}

	res[i*blockDim.x + j] = sum;

}


// write kernels here...
__global__ void a_plus_bt(int *matrixA, int *matrixBT,int p,int q)
{
	extern __shared__ int f[];
	f[threadIdx.x] = matrixA[blockIdx.x*blockDim.x + threadIdx.x];
	matrixA[blockIdx.x*blockDim.x + threadIdx.x] = f[threadIdx.x] + matrixBT[blockIdx.x*blockDim.x + threadIdx.x];

}

__global__ void transpose(int *mat, int *res)
{
res[threadIdx.x*gridDim.x + blockIdx.x]  =  mat[blockIdx.x*blockDim.x+threadIdx.x];
}
// function to compute the output matrix
void compute(int p, int q, int r, int s, int *h_matrixA, int *h_matrixB, 
	         int *h_matrixC, int *h_matrixD, int *h_matrixX) {
	// variable declarations...
	int *matrixA, *matrixB, *matrixC, *matrixD;
	int *matrixBT, *matrixDT;
    int *cdtt ; // for storing (C D^T)^T
    int *res;
	// allocate memory...
	hipMalloc(&matrixA, p*q*sizeof(int));
	hipMalloc(&matrixB, q*p*sizeof(int));
	hipMalloc(&matrixC, q*r*sizeof(int));
	hipMalloc(&matrixD, s*r*sizeof(int));
	hipMalloc(&matrixBT, p*q*sizeof(int));
	hipMalloc(&matrixDT, r*s*sizeof(int));
	hipMalloc(&cdtt, s*q*sizeof(int));
	hipMalloc(&res, p*s*sizeof(int));

	// copy the values...
	hipMemcpy(matrixA, h_matrixA, p*q*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(matrixB, h_matrixB, q*p*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(matrixC, h_matrixC, q*r*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(matrixD, h_matrixD, s*r*sizeof(int), hipMemcpyHostToDevice);

	// call the kernels for doing required computations...
	transpose<<<q,p>>>(matrixB, matrixBT);
	//transpose<<<s,r>>>(matrixD, matrixDT);//to increase memory cohelesing do not transpose matrix D
	int *matrixCDT;
	hipMalloc(&matrixCDT, q*s*sizeof(int));

	matmul<<<q,s>>>(matrixC, matrixD,matrixCDT,r);

	a_plus_bt<<<p,q, q*sizeof(int)>>>(matrixA, matrixBT,p,q);// After this kernel execution matrixA have (A+B^T)
	hipFree(matrixC);
	hipFree(matrixB);
	hipFree(matrixD);
	transpose<<<q,s>>>(matrixCDT, cdtt);
	matmul<<<p,s>>>(matrixA, cdtt, res, q);
	// copy the result back...
	hipMemcpy(h_matrixX, res,p*s*sizeof(int), hipMemcpyDeviceToHost);
	
	// deallocate the memory...
	hipFree(cdtt);
	hipFree(matrixCDT);
	hipFree(matrixA);
}

// function to read the input matrices from the input file
void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}

// function to write the output matrix into the output file
void writeMatrix(FILE *outputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fprintf(outputFilePtr, "%d ", matrix[i*cols+j]);
		}
		fprintf(outputFilePtr, "\n");
	}
}

int main(int argc, char **argv) {
	// variable declarations
	int p, q, r, s;
	int *matrixA, *matrixB, *matrixC, *matrixD, *matrixX;
	struct timeval t1, t2;
	double seconds, microSeconds;

	// get file names from command line
	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	// file pointers
	FILE *inputFilePtr, *outputFilePtr;
    
    inputFilePtr = fopen(inputFileName, "r");
	if(inputFilePtr == NULL) {
	    printf("Failed to open the input file.!!\n"); 
		return 0;
	}

	// read input values
	fscanf(inputFilePtr, "%d %d %d %d", &p, &q, &r, &s);

	// allocate memory and read input matrices
	matrixA = (int*) malloc(p * q * sizeof(int));
	matrixB = (int*) malloc(q * p * sizeof(int));
	matrixC = (int*) malloc(q * r * sizeof(int));
	matrixD = (int*) malloc(s * r * sizeof(int));
	readMatrix(inputFilePtr, matrixA, p, q);
	readMatrix(inputFilePtr, matrixB, q, p);
	readMatrix(inputFilePtr, matrixC, q, r);
	readMatrix(inputFilePtr, matrixD, s, r);

	// allocate memory for output matrix
	matrixX = (int*) malloc(p * s * sizeof(int));

	// call compute function to get the output matrix. it is expected that 
	// the compute function will store the result in matrixX.
	gettimeofday(&t1, NULL);
	compute(p, q, r, s, matrixA, matrixB, matrixC, matrixD, matrixX);
	hipDeviceSynchronize();
	gettimeofday(&t2, NULL);

	// print the time taken by the compute function
	seconds = t2.tv_sec - t1.tv_sec;
	microSeconds = t2.tv_usec - t1.tv_usec;
	printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);

	// store the result into the output file
	outputFilePtr = fopen(outputFileName, "w");
	writeMatrix(outputFilePtr, matrixX, p, s);

	// close files
    fclose(inputFilePtr);
    fclose(outputFilePtr);

	// deallocate memory
	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixD);
	free(matrixX);

	return 0;
}